#include "hip/hip_runtime.h"
// https://koreatechackr-my.sharepoint.com/:b:/g/personal/bluekds_koreatech_ac_kr/EWq2qFt0rJtBjFaYymZ81FEBOpVF_Z9YPmNBHuyn-ol4WA?e=106gGk
// Ref: [선형 인덱스 <-> 다차원 인덱스 변환](https://blog.naver.com/ipsy2003/221853617999)

#include <iostream>
#include <random>
#include <cstdlib>

#include "DS_timer.h"

const float epsilon = 1e-3;

bool is_equivalent(float a, float b) {
    return abs(a - b) < epsilon;
}

enum TIMER_NAMES {
    CPU_SERIAL,
    CPU_PARALLEL,
    GPU,
    GPU_HOST_TO_DEVICE,
    GPU_COMPUTAION,
    GPU_DEVICE_TO_HOST,
    NUM_TIMERS
};

__global__ void cuda_matrix_mult(float* A, float* B, float* C, int N, int M, int L) {
    // C에서의 선형 인덱스.
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // 범위 검사.
    if (index >= N * L)
        return;

    // C에서의 이차원 인덱스.
    size_t i = index / L;
    size_t j = index % L;

    // C에서의 각 요소에 대응하는 A, B의 요소 처리.
    for (size_t k = 0; k < M; k++)
        C[index] += A[i * M + k] + B[k * L + j];
}

int main() {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    DS_timer timer(NUM_TIMERS);

    timer.setTimerName(CPU_SERIAL        , (char*)"Serial             ");
    timer.setTimerName(CPU_PARALLEL      , (char*)"Parallel           ");

    timer.setTimerName(GPU               , (char*)"GPU                ");
    timer.setTimerName(GPU_HOST_TO_DEVICE, (char*)"GPU: Host -> Device");
    timer.setTimerName(GPU_COMPUTAION    , (char*)"GPU: Computation   ");
    timer.setTimerName(GPU_DEVICE_TO_HOST, (char*)"GPU: Device -> Host");

    const int N = 512;
    const int M = 2048;
    const int L = 1024;

    float* A = new float[N * M];
    float* B = new float[M * L];
    float* C = new float[N * L];

    for (int i = 0; i < N; i++)
        for (int j = 0; j < M; j++)
            A[i * M + j] = dist(gen);

    for (int i = 0; i < M; i++)
        for (int j = 0; j < L; j++)
            B[i * L + j] = dist(gen);

    memset(C, 0, N * L * sizeof(float));

    // Serial.
    timer.onTimer(CPU_SERIAL);

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            for (int k = 0; k < M; k++)
                C[i * L + j] += A[i * M + k] + B[k * L + j];

    timer.offTimer(CPU_SERIAL);

    // Parallel.
    float* C_parallel = new float[N * L];

    memset(C_parallel, 0, N * L * sizeof(float));

    timer.onTimer(CPU_PARALLEL);

    #pragma omp parallel for
    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            for (int k = 0; k < M; k++) {
                #pragma omp atomic
                C_parallel[i * L + j] += A[i * M + k] + B[k * L + j];
            }

    timer.offTimer(CPU_PARALLEL);

    // GPU.
    float* A_gpu, *B_gpu, *C_gpu, *temp_gpu;

    hipMalloc(&A_gpu   , N * M * sizeof(float));
    hipMalloc(&B_gpu   , M * L * sizeof(float));
    hipMalloc(&temp_gpu, N * L * sizeof(float));
    C_gpu = new float[N * L];

    hipMemset(temp_gpu, 0, N * L * sizeof(float));

    timer.onTimer(GPU);

    timer.onTimer(GPU_HOST_TO_DEVICE);
    hipMemcpy(A_gpu, A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, M * L * sizeof(float), hipMemcpyHostToDevice);
    timer.offTimer(GPU_HOST_TO_DEVICE);

    timer.onTimer(GPU_COMPUTAION);
    size_t total_size = N * L;
    size_t unit_size = 256;

    dim3 grid_dim(ceil((float)total_size / unit_size));
    dim3 block_dim(unit_size);

    cuda_matrix_mult<<<grid_dim, block_dim>>>(A_gpu, B_gpu, temp_gpu, N, M, L);
    hipDeviceSynchronize();
    timer.offTimer(GPU_COMPUTAION);

    timer.onTimer(GPU_DEVICE_TO_HOST);
    hipMemcpy(C_gpu, temp_gpu, N * L * sizeof(float), hipMemcpyDeviceToHost);
    timer.offTimer(GPU_DEVICE_TO_HOST);

    timer.offTimer(GPU);

    // Checking.
    bool is_correct_parallel = true;
    bool is_correct_gpu = true;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            if (!is_equivalent(C[i * L + j], C_parallel[i * L + j])) {
                is_correct_parallel = false;
                goto loop_parallel;
            }
    loop_parallel:

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            if (!is_equivalent(C[i * L + j], C_gpu[i * L + j])) {
                is_correct_gpu = false;
                goto loop_gpu;
            }
    loop_gpu:

    std::cout << "Epsilon:  " << epsilon << "\n";
    std::cout << "Parallel: " << (is_correct_parallel ? "Succeeded" : "Failed") << "\n";
    std::cout << "GPU:      " << (is_correct_gpu      ? "Succeeded" : "Failed") << "\n";

    timer.printTimer();

    delete[] A, B, C;
    delete[] C_parallel, C_gpu;

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(temp_gpu);

    return 0;
}