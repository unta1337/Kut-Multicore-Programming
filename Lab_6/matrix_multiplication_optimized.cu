#include "hip/hip_runtime.h"
// https://koreatechackr-my.sharepoint.com/:b:/g/personal/bluekds_koreatech_ac_kr/EY6KGQhO465FsjcSb1eBQV8BPrua8wKU8sGCkvrEgZz0Ig?e=xViTHA

// 시도한 최적화:
//  1. row -> threadIdx.x, col -> threadIdx.y로 하던 것을 반대로 row -> threadIdx.y, col -> threadIdx.x로 변경.
//  2. 각 쓰레드가 적절히 분배되어 적당한 Warp을 구성하도록 Block Dim 조정.
//  3. A와 B에 대한 Shared Memory를 하나의 변수에 할당하여 연속된 공간에 할당됨을 보장하여 메모리 뱅크 관리.
//  4. 각 쓰레드가 적절히 분배되어 서로 다른 메모리 뱅크를 참조하도록 Block Dim 조정.
//  5. A에 대한 Shared Memory에 각 쓰레드가 서로 다른 메모리 뱅크를 참조하도록 인덱싱 방법 수정.
//  6. 자주 사용하는 변수에 대한 CUDA 변수를 로컬 레지스터에 저장. (threadIdx.x 등.)
//  7. for문의 종료 조건에 대한 식을 로컬 레지스터에 저장.
//  8. 암묵적인 형변환에 대한 오버헤드를 줄이기 위해 모든 정수형 변수를 unsigned int로 통일.
//  9. 암묵적인 형변환에 대한 오버헤드를 줄이기 위해 나눗셈에 사용되는 define 상수를 float 형으로 변경.
// 10. 조건문을 간단화하여 연산 비용 감소.

#include <iostream>
#include <random>
#include <cstdlib>

#include "DS_timer.h"

#define N 2048
#define M 4096
#define L 1024

#define UNIT 16

#define F_N 2048.0f
#define F_M 4096.0f
#define F_L 1024.0f

#define F_UNIT 16.0f

const float epsilon = 1e-3;

bool is_equivalent(float a, float b) {
    return abs(a - b) < epsilon;
}

enum TIMER_NAMES {
    CPU_SERIAL,
    CPU_PARALLEL,

    GPU,
    GPU_HOST_TO_DEVICE,
    GPU_COMPUTAION,
    GPU_DEVICE_TO_HOST,

    GPU_SHARED,
    GPU_SHARED_HOST_TO_DEVICE,
    GPU_SHARED_COMPUTAION,
    GPU_SHARED_DEVICE_TO_HOST,

    NUM_TIMERS
};

__global__ void cuda_matrix_mult(float* A, float* B, float* C) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N * L)
        return;

    size_t i = index / L;
    size_t j = index % L;

    float result = 0.0f;
    for (size_t k = 0; k < M; k++)
        result += A[i * M + k] * B[k * L + j];

    C[index] = result;
}

__global__ void cuda_matrix_mult_shared(float* __A, float* __B, float* C) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int i_local = threadIdx.y;
    unsigned int j_local = threadIdx.x;

    __shared__ float mem_chunk[2][UNIT][UNIT];

    float result = 0.0f;
    unsigned int iter = (unsigned int)(F_M / F_UNIT) + 1U;

    for (unsigned int t = 0; t < iter; t++) {
        unsigned int offset = t * UNIT;

        mem_chunk[0][j_local][i_local] = __A[i * M + (j_local + offset)] * (j_local + offset < M);
        mem_chunk[1][i_local][j_local] = __B[(i_local + offset) * L + j] * (i_local + offset < M);

        __syncthreads();

        for (unsigned int k = 0; k < UNIT; k++) {
            result += mem_chunk[0][k][i_local] * mem_chunk[1][k][j_local];
        }

        __syncthreads();
    }

    if (i >= N || j >= L)
        return;

    C[i * L + j] = result;
}

int main() {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    DS_timer timer(NUM_TIMERS);

    timer.setTimerName(CPU_SERIAL               , (char*)"Serial                    ");
    timer.setTimerName(CPU_PARALLEL             , (char*)"Parallel                  ");

    timer.setTimerName(GPU                      , (char*)"GPU                       ");
    timer.setTimerName(GPU_HOST_TO_DEVICE       , (char*)"GPU: Host -> Device       ");
    timer.setTimerName(GPU_COMPUTAION           , (char*)"GPU: Computation          ");
    timer.setTimerName(GPU_DEVICE_TO_HOST       , (char*)"GPU: Device -> Host       ");

    timer.setTimerName(GPU_SHARED               , (char*)"GPU Shared                ");
    timer.setTimerName(GPU_SHARED_HOST_TO_DEVICE, (char*)"GPU Shared: Host -> Device");
    timer.setTimerName(GPU_SHARED_COMPUTAION    , (char*)"GPU Shared: Computation   ");
    timer.setTimerName(GPU_SHARED_DEVICE_TO_HOST, (char*)"GPU Shared: Device -> Host");

    float* A = new float[N * M];
    float* B = new float[M * L];
    float* C = new float[N * L];

    for (int i = 0; i < N; i++)
        for (int j = 0; j < M; j++)
            A[i * M + j] = dist(gen);

    for (int i = 0; i < M; i++)
        for (int j = 0; j < L; j++)
            B[i * L + j] = dist(gen);

    memset(C, 0, N * L * sizeof(float));

    // Serial.
    timer.onTimer(CPU_SERIAL);

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            for (int k = 0; k < M; k++)
                C[i * L + j] += A[i * M + k] * B[k * L + j];

    timer.offTimer(CPU_SERIAL);

    // Parallel.
    float* C_parallel = new float[N * L];

    memset(C_parallel, 0, N * L * sizeof(float));

    timer.onTimer(CPU_PARALLEL);

    #pragma omp parallel for
    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            for (int k = 0; k < M; k++) {
                #pragma omp atomic
                C_parallel[i * L + j] += A[i * M + k] * B[k * L + j];
            }

    timer.offTimer(CPU_PARALLEL);

    // GPU.
    float* A_gpu, *B_gpu, *C_gpu, *temp_gpu;

    hipMalloc(&A_gpu   , N * M * sizeof(float));
    hipMalloc(&B_gpu   , M * L * sizeof(float));
    hipMalloc(&temp_gpu, N * L * sizeof(float));
    C_gpu = new float[N * L];

    hipMemset(temp_gpu, 0, N * L * sizeof(float));

    timer.onTimer(GPU);

    timer.onTimer(GPU_HOST_TO_DEVICE);
    hipMemcpy(A_gpu, A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, M * L * sizeof(float), hipMemcpyHostToDevice);
    timer.offTimer(GPU_HOST_TO_DEVICE);

    timer.onTimer(GPU_COMPUTAION);
    size_t total_size = N * L;
    size_t unit_size = 256;

    dim3 grid_dim(ceil((float)total_size / unit_size));
    dim3 block_dim(unit_size);

    cuda_matrix_mult<<<grid_dim, block_dim>>>(A_gpu, B_gpu, temp_gpu);
    hipDeviceSynchronize();
    timer.offTimer(GPU_COMPUTAION);

    timer.onTimer(GPU_DEVICE_TO_HOST);
    hipMemcpy(C_gpu, temp_gpu, N * L * sizeof(float), hipMemcpyDeviceToHost);
    timer.offTimer(GPU_DEVICE_TO_HOST);

    timer.offTimer(GPU);

    // GPU Shared,
    float* A_gpu_shared, *B_gpu_shared, *C_gpu_shared, *temp_gpu_shared;

    hipMalloc(&A_gpu_shared   , N * M * sizeof(float));
    hipMalloc(&B_gpu_shared   , M * L * sizeof(float));
    hipMalloc(&temp_gpu_shared, N * L * sizeof(float));
    C_gpu_shared = new float[N * L];

    hipMemset(temp_gpu_shared, 0, N * L * sizeof(float));

    timer.onTimer(GPU_SHARED);

    timer.onTimer(GPU_SHARED_HOST_TO_DEVICE);
    hipMemcpy(A_gpu_shared, A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu_shared, B, M * L * sizeof(float), hipMemcpyHostToDevice);
    timer.offTimer(GPU_SHARED_HOST_TO_DEVICE);

    timer.onTimer(GPU_SHARED_COMPUTAION);
    dim3 grid_dim_shared(ceil((float)L / UNIT), ceil((float)N / UNIT));
    dim3 block_dim_shared(UNIT, UNIT);

    cuda_matrix_mult_shared<<<grid_dim_shared, block_dim_shared>>>(A_gpu_shared, B_gpu_shared, temp_gpu_shared);
    hipDeviceSynchronize();
    timer.offTimer(GPU_SHARED_COMPUTAION);

    timer.onTimer(GPU_SHARED_DEVICE_TO_HOST);
    hipMemcpy(C_gpu_shared, temp_gpu_shared, N * L * sizeof(float), hipMemcpyDeviceToHost);
    timer.offTimer(GPU_SHARED_DEVICE_TO_HOST);

    timer.offTimer(GPU_SHARED);

    // Checking.
    bool is_correct_parallel = true;
    bool is_correct_gpu = true;
    bool is_correct_gpu_shared = true;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            if (!is_equivalent(C[i * L + j], C_parallel[i * L + j])) {
                is_correct_parallel = false;
                goto loop_parallel;
            }
    loop_parallel:

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            if (!is_equivalent(C[i * L + j], C_gpu[i * L + j])) {
                is_correct_gpu = false;
                goto loop_gpu;
            }
    loop_gpu:

    for (int i = 0; i < N; i++)
        for (int j = 0; j < L; j++)
            if (!is_equivalent(C[i * L + j], C_gpu_shared[i * L + j])) {
                is_correct_gpu_shared = false;
                goto loop_gpu_shared;
            }
    loop_gpu_shared:

    std::cout << "Epsilon:    " << epsilon << "\n";
    std::cout << "Parallel:   " << (is_correct_parallel   ? "Succeeded" : "Failed") << "\n";
    std::cout << "GPU:        " << (is_correct_gpu        ? "Succeeded" : "Failed") << "\n";
    std::cout << "GPU Shared: " << (is_correct_gpu_shared ? "Succeeded" : "Failed") << "\n";

    timer.printTimer();

    delete[] A, B, C;
    delete[] C_parallel, C_gpu, C_gpu_shared;

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(temp_gpu);

    hipFree(A_gpu_shared);
    hipFree(B_gpu_shared);
    hipFree(temp_gpu_shared);

    return 0;
}