#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>

#include "DS_timer.h"

#define F(x) ((x) * (x))

const double epsilon = 1e-3;

bool is_equivalent(double a, double b) {
    return abs(a - b) < epsilon;
}

enum TIMER_NAMES {
    CPU_SERIAL,
    CPU_PARALLEL,

    GPU,
    GPU_HOST_TO_DEVICE,
    GPU_COMPUTAION,
    GPU_DEVICE_TO_HOST,

    NUM_TIMERS
};

__global__ void calc_definite_integral(int a, int b, int n, double* I_chunks) {
    size_t beta = gridDim.x;
    size_t alpha = n / beta;

    size_t block_begin = alpha * blockIdx.x;
    bool is_end = blockIdx.x == beta - 1;
    size_t block_end = !is_end * (alpha * (blockIdx.x + 1)) +
                        is_end * n;

    double dx = (double)(b - a) / n;
    __shared__ double local_sum[1024];

    for (size_t i = block_begin + threadIdx.x; i < block_end; i += blockDim.x) {
        double x = a + dx * i;
        local_sum[threadIdx.x] += (F(x) + F(x + dx)) * dx / 2;
    }

    __syncthreads();

    for (size_t stride = 1; stride < blockDim.x; stride <<= 1) {
		if (threadIdx.x % (2 * stride) == 0)
			local_sum[threadIdx.x] += local_sum[threadIdx.x + stride];
		__syncthreads();
	}

    if (threadIdx.x == 0)
        I_chunks[blockIdx.x] += local_sum[0];
}

int main(int argc, char* argv[]) {
    int a, b, n;

    if (argc == 1) {
        a = 0;
        b = 1024;
        n = 1073741824;
    } else if (argc == 4) {
        a = strtol(argv[1], NULL, 10);
        b = strtol(argv[2], NULL, 10);
        n = strtol(argv[3], NULL, 10);
    } else {
        fprintf(stderr, "Argument not enough: a b n.\n");
        return 1;
    }

    DS_timer timer(NUM_TIMERS);

    timer.setTimerName(CPU_SERIAL        , (char*)"Serial             ");
    timer.setTimerName(CPU_PARALLEL      , (char*)"Parallel           ");

    timer.setTimerName(GPU               , (char*)"GPU                ");
    timer.setTimerName(GPU_HOST_TO_DEVICE, (char*)"GPU: Host -> Device");
    timer.setTimerName(GPU_COMPUTAION    , (char*)"GPU: Computation   ");
    timer.setTimerName(GPU_DEVICE_TO_HOST, (char*)"GPU: Device -> Host");

    // Serial.
    timer.onTimer(CPU_SERIAL);

    double dx = (double)(b - a) / n;

    double I = 0.0f;
    for (size_t i = 0; i < n; i++) {
        double x = a + dx * i;
        I += (F(x) + F(x + dx)) * dx / 2;
    }

    timer.offTimer(CPU_SERIAL);

    // Parallel.
    timer.onTimer(CPU_PARALLEL);

    double p_dx = (double)(b - a) / n;
    double p_I = 0.0;

    #pragma omp parallel reduction(+ : p_I)
    {
        size_t index = omp_get_thread_num();

        #pragma omp for
        for (int i = 0; i < n; i++) {
            double x = a + p_dx * i;
            p_I += (F(x) + F(x + p_dx)) * p_dx / 2;
        }
    }

    timer.offTimer(CPU_PARALLEL);

    // GPU.
    // Setup.
    size_t num_chunks = 20;
    size_t unit = 1024;

    double d_I = 0.0;

    double* I_chunks = new double[num_chunks];
    double* d_I_chunks;
    hipMalloc(&d_I_chunks, num_chunks * sizeof(double));

    timer.onTimer(GPU);

    timer.onTimer(GPU_HOST_TO_DEVICE);
    hipMemset(d_I_chunks, 0, num_chunks * sizeof(double));
    timer.offTimer(GPU_HOST_TO_DEVICE);

    timer.onTimer(GPU_COMPUTAION);
    calc_definite_integral<<<num_chunks, unit>>>(a, b, n, d_I_chunks);
    hipDeviceSynchronize();
    timer.offTimer(GPU_COMPUTAION);

    timer.onTimer(GPU_DEVICE_TO_HOST);
    hipMemcpy(I_chunks, d_I_chunks, num_chunks * sizeof(double), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < num_chunks; i++)
        d_I += I_chunks[i];
    timer.offTimer(GPU_DEVICE_TO_HOST);

    timer.offTimer(GPU);

    printf("(a, b): (%d, %d)\n", a, b);
    printf("n: %d\n\n", n);

    printf("CPU Serial  : %f\n",     I);
    printf("CPU Parallel: %f\n",   p_I);
    printf("GPU:          %f\n\n", d_I);

    printf("CPU Parallel: %s.\n", is_equivalent(I, p_I) ? "Succeded" : "Failed");
    printf("GPU         : %s.\n", is_equivalent(I, d_I) ? "Succeded" : "Failed");

    timer.printTimer();

    return 0;
}
